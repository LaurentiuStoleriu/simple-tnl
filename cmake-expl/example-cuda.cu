#include <iostream>
#include <random>
#include <TNL/Containers/Vector.h>
#include <TNL/Containers/StaticArray.h>
#include <TNL/Algorithms/parallelFor.h>

using namespace TNL;
using namespace TNL::Containers;
using namespace TNL::Algorithms;

//std::random_device rd;
//std::mt19937 gen(rd());
//std::uniform_real_distribution<> dis(100.0, 200.0);

template<typename Device>
void initMeshFunction (const int xSize, const int ySize, Vector<double, Device> &v, const double &c)
{
    auto view = v.getView();
    auto init = [=] __cuda_callable__ (const StaticArray<2, int> &i) mutable
    {
        view[i.y() * xSize + i.x()] = c;
    };
    StaticArray<2, int> begin{0, 0, 0};
    StaticArray<2, int> end{xSize, ySize};
    parallelFor<Device>(begin, end, init);
}

int main(int argc, char* argv[])
{
    const int xSize(10000), ySize(10000);
    const int size = xSize * ySize;

    Vector<double, Devices::Host> host_v(size);
    initMeshFunction(xSize, ySize, host_v, 1.0); 
    std::cout << "DONE!!" << std::endl;

#ifdef __HIPCC__
    Vector<double, Devices::Cuda> cuda_v(size);
    initMeshFunction(xSize, ySize, cuda_v, 1.0);
    std::cout << "DONE CUDA!!" << std::endl;
#endif
    return EXIT_SUCCESS;
}