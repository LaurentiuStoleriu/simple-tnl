#include "hip/hip_runtime.h"
#include <iostream>
#include <TNL/FileName.h>
#include <TNL/Containers/Vector.h>
//#include <TNL/Solvers/ODE/Euler.h>
#include <TNL/Solvers/ODE/Merson.h>
////////////////////
//#include "write.h"
#include <fstream>

template<typename Vector>
void write( std::fstream& file, const Vector& u, const int n, const double& h, const double& time )
{
    std::cout/*file*/ << "# time = " << time << std::endl;
    for( int i = 0; i < n; i++ )
        std::cout/*file*/ << i*h << " " << u.getElement( i ) << std::endl;
    std::cout/*file*/ << std::endl;
}
/////////////////////

//using Real = double;
//using Index = int;

template< typename Device >
void solveHeatEquation( const char* file_name )
{
    using Vector = TNL::Containers::Vector< double, Device, int >;
    using VectorView = typename Vector::ViewType;
    //using ODESolver = TNL::Solvers::ODE::Euler< Vector >;
    using ODESolver = TNL::Solvers::ODE::Merson< Vector >;

/***
* Parameters of the discretisation
*/
    const double final_t = 0.05;
    const double output_time_step = 0.005;
    const int n = 41;
    const double h = 1.0 / ( n - 1 );
    const double tau = 0.1 * h * h;
    const double h_sqr_inv = 1.0 / ( h * h );

/***
* Initial condition
*/
    Vector u( n );
    u.forAllElements( [=] __cuda_callable__ ( int i, double& value ) {
    const double x = i * h;
    if( x >= 0.4 && x <= 0.6 )
        value = 1.0;
    else 
        value = 0.0;
    } );

    std::fstream file;
    file.open( file_name, std::ios::out );
    write( file, u, n, h, (double) 0.0 );

/***
* Setup of the solver
*/
    ODESolver solver;
    solver.setTau(  tau );
    solver.setTime( 0.0 );

/***
* Time loop
*/
    while( solver.getTime() < final_t )
    {
        solver.setStopTime( TNL::min( solver.getTime() + output_time_step, final_t ) );
        auto f = [=] __cuda_callable__ ( int i, const VectorView& u, VectorView& fu ) mutable {
         if( i == 0 || i == n-1 )                // boundary nodes -> boundary conditions
            fu[ i ] = 0.0;
         else                                    // interior nodes -> approximation of the second derivative
            fu[ i ] = h_sqr_inv * (  u[ i - 1 ] - 2.0 * u[ i ] + u[ i + 1 ] );
        };
        auto time_stepping = [=] ( const double& t, const double& tau, const VectorView& u, VectorView& fu ) {
            TNL::Algorithms::parallelFor< Device >( 0, n, f, u, fu ); };
            solver.solve( u, time_stepping );
        write( file, u, n, h, solver.getTime() ); // write the current state to a file
    }
}

int main( int argc, char* argv[] )
{
    //TNL::String file_name( argv[ 1 ] );
    TNL::String file_name("/home/lali/TITAN-ROG-sync/CUDA/simple-tnl/ODE/ODESolver-HeatEquationExample-result.out");

    std::cout << file_name.getString();

    //solveHeatEquation< TNL::Devices::Host >( file_name.getString() );
#ifdef __HIPCC__
    solveHeatEquation< TNL::Devices::Cuda >( file_name.getString() );
#endif
}